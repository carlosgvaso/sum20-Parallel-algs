#include "hip/hip_runtime.h"
/** Homework 3 question 1 code
 *
 * \file q1.cu
 * \author Jose Carlos Martinez Garcia-Vaso <carlosgvaso@utexas.edu>
 * \author Utkarsh Vardan <uvardan@utexas.edu>
 */


#include <cstdio>			// standard I/O
#include <string>			// strings
#include <fstream>			// streams
#include <vector>			// std::vector
#include <sstream>			// std::stringstream
#include <hip/hip_runtime.h>	// CUDA functions


// Globals
#define DEBUG 1	//! Enable debug messages (0: no messages, 1: some messages, 2: all messages)

#define INPUT_FILE "inp.txt"		//! Input filename
#define OUTPUT_FILE_Q1A "q1a.txt"	//! Q1 a output filename
#define OUTPUT_FILE_Q1B "q1b.txt"	//! Q1 b output filename

#define EXIT_OK 0		//! Exit code success
#define EXIT_FATAL 1	//! Exit code unrecoverable error


/** Read input from file
 *
 * This function assumes the file contains a single line, as per the format in
 * the README.txt.
 *
 * \param	filename	Name of input file to read
 * \return	Vector containing the input array in the file
 */
std::vector<int> read_input (std::string filename) {
	// Create a vector of integers to store the array in file
	std::vector<int> v_in;

	// Create an input filestream
	std::ifstream fin(filename);

	// Make sure the file is open
	if (!fin.is_open()) {
		fprintf(stderr, "ERROR:read_input: Could not open file\n");
		exit(EXIT_FATAL);
	}

	// Helper vars
	std::string line;
	int val;

	// Read the column names
	if (fin.good()) {
		// Extract the first line in the file
		std::getline(fin, line);

		// Create a stringstream from line
		std::stringstream ss(line);

		// Extract each integer
		while (ss >> val) {

			// Add the current integer to the vector
			v_in.push_back(val);

			// If the next token is a comma, ignore it and move on
			if (ss.peek() == ',') ss.ignore();
		}
	}

	// Close file
	fin.close();

	return v_in;
}

/** Write formated output to file
 *
 * This function uses the output format described in the README.txt file.
 *
 * \param	filename	Name of the output file
 * \param	v_out		Vector to save to file
 */
void write_output (std::string filename, const std::vector<int> &v_out) {
	// Create an output filestream object
	std::ofstream fout(filename);

	// Send v_out vector entries to the stream
	for (int i = 0; i < v_out.size(); ++i) {
		fout << v_out.at(i);
		if(i != v_out.size() - 1) fout << ", "; // No comma at end of line
	}
	//fout << "\n";	// inp.txt doesn't have a newline at the end of the file

	// Close the file
	fout.close();
}

/**
 *
 */
__global__ void parallelScanMinKernel(int *d_out, int *d_in, int n)
{
	int gid = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	int val = 0;

	// Ensure we only access available array entries
	if (gid < n) {
		#if DEBUG >= 2
		if (tid == 0) {
			printf("\tIterations:\n\t\tBlock %d: d = %d: d_in = [ ",
				   blockIdx.x, 0);
			for (int i=0; i<n; ++i) {
				if (i == n-1) {
					printf("%d ]\n", d_in[i]);
				} else {
					printf("%d, ", d_in[i]);
				}
			}
		}
		#endif
		
		for (int d=1; d<n; d=d*2) {
			if (gid >= d) {
				val = d_in[gid - d];
			}
			__syncthreads();

			if (gid >= d) {
				d_in[gid] = d_in[gid] <= val ? d_in[gid] : val;
			}
			__syncthreads();

			#if DEBUG >= 2
			if (tid == 0) {
				printf("\t\tBlock %d: d = %d: d_in = [ ", blockIdx.x, d);
				for (int i=0; i<n; ++i) {
					if (i == n-1) {
						printf("%d ]\n", d_in[i]);
					} else {
						printf("%d, ", d_in[i]);
					}
				}
			}
			#endif
		}

		/* The result for a block is in the last thread entry for that block.
		 * If n is not a multiple of blockDim.x, the result is the entry of
		 * gid == n-1.
		 */
		if ((tid == blockDim.x-1 && gid != n-1) || gid == n-1) {
			d_out[blockIdx.x] = d_in[gid];

			#if DEBUG
			printf("\t\tBlock %d min: d_out[%d] = %d\n",
				   blockIdx.x, blockIdx.x, d_out[blockIdx.x]);
			#endif
		}
	}
}

/** Q1 a) Compute minA, the minimum value in the input array
 *
 * This function uses the Hillis-Steele version of parallel scan to find the
 * minimum value in the input array. Then, it outputs the result to the
 * OUTPUT_FILE_Q1A output file.
 *
 * This function will only work for problems of size (input array size)
 * (hipDeviceProp_t.maxThreadsPerBlock)^2. For example, if we have a
 * hipDeviceProp_t.maxThreadsPerBlock = 1024 (a normal value for current Nvidia
 * GPUs), the max problem size is N = 1024^2 = 1,048,576. Since the professor
 * said the max graded size should be 10^6, this restriction sufices.
 *
 * \param	v_in		Input array as a vector
 * \param	dev_props	CUDA device properties
 */
void q1a (const std::vector<int> &v_in, hipDeviceProp_t *dev_props) {
	#if DEBUG
	printf("\tTransfering input array to GPU memory...\n");
	#endif

	// Declare GPU memory pointers
	int *d_in, *d_intermediate, *d_out;

	// Allocate GPU memory
	int N = v_in.size();				// Problem size (input array size)
	int d_in_size = N * sizeof(int);	// Input array size in bytes
	int d_out_size = sizeof(int);		// Output array size in bytes

	#if DEBUG
	printf("\tN (input array size): %d\n", N);
	#endif

	if (N > ((int)((*dev_props).maxThreadsPerBlock) * (int)((*dev_props).maxThreadsPerBlock))) {
		fprintf(stderr, "ERROR:q1a: problem size (input array size) is too large\n");
		exit(EXIT_FATAL);
	}

	hipMalloc((void **) &d_in, d_in_size);
	hipMalloc((void **) &d_intermediate, d_in_size); // overallocated
	hipMalloc((void **) &d_out, d_out_size);

	/* Transfer the input array to the GPU
	 * Since the elements of a vector are stored contiguously in memory, we can
	 * pass a pointer to the first element of the vector, and that will act as
	 * if we passed a C array.
	 */
	hipMemcpy(d_in, &v_in[0], d_in_size, hipMemcpyHostToDevice);

	#if DEBUG
	// Set up a timer to measure the elapsed time to find the min
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("\tFinding minimum entry in the array...\n");
	hipEventRecord(start, 0);
	#endif

	// Calculate the number of blocks and threads to use
	int threads_per_block = (int)((*dev_props).maxThreadsPerBlock); // Max number of threads per block
	int blocks_per_grid = (N + (threads_per_block - 1)) / threads_per_block;

	#if DEBUG
	printf("\tThreads per block: %d\n", threads_per_block);
	printf("\tBlocks per grid: %d\n", blocks_per_grid);
	printf("\tRunning kernel...\n");
	#endif

	// Launch the kernel to find min
	parallelScanMinKernel<<<blocks_per_grid, threads_per_block>>>
		(d_intermediate, d_in, N);
	
	// Make sure all the blocks finish executing
	hipDeviceSynchronize();
	hipDeviceSynchronize();

	// If there are more than one block, we need to repeat the process with their results
	if (blocks_per_grid > 1) {
		#if DEBUG
		// Copy array to host
		int *a_out;
		a_out = (int*) malloc(d_in_size);
		hipMemcpy(a_out, d_intermediate, d_in_size, hipMemcpyDeviceToHost);

		printf("\tBlock results: d_intermediate = [ ");
		for (int i=0; i<blocks_per_grid; ++i) {
			if (i == blocks_per_grid-1) {
				printf("%d ]\n", a_out[i]);
			} else {
				printf("%d, ", a_out[i]);
			}
		}
		free(a_out);
		#endif

		#if DEBUG
		printf("\tThreads per block: %d\n", blocks_per_grid);
		printf("\tBlocks per grid: %d\n", 1);
		printf("\tRunning kernel...\n");
		#endif

		// Fill one block with the results from the other blocks
		parallelScanMinKernel<<<1, blocks_per_grid>>>
			(d_out, d_intermediate, blocks_per_grid);
	}
	
	#if DEBUG
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Calculate elapsed time, and print it
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\tAverage time elapsed: %f\n", elapsedTime);
	#endif

	// Copy back the min result from GPU
	int a_out;
	if (blocks_per_grid > 1) {
		hipMemcpy(&a_out, d_out, d_out_size, hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(&a_out, d_intermediate, d_out_size, hipMemcpyDeviceToHost);
	}

	#if DEBUG
	printf("\tMin: %d\n", a_out);
	#endif
	
	// Copy resulting array to output vector
	std::vector<int> v_out;
	v_out.push_back(a_out);

	// Free GPU memory
	hipFree(d_in);
	hipFree(d_intermediate);
	hipFree(d_out);

	write_output(OUTPUT_FILE_Q1A, v_out);
}

/** Q1 b) Compute an array B such that B[i] is the last digit of A[i] for all i
 *
 * \param	v_in		Input array as a vector
 * \param	dev_props	CUDA device properties
 */
 void q1b (const std::vector<int> &v_in, hipDeviceProp_t *dev_props) {
	std::vector<int> v_out;

	// TODO: Implement
	#if DEBUG
	printf("\tThreads per block: %d\n",
		   (int)((*dev_props).maxThreadsPerBlock));
	#endif

	v_out = v_in;

	write_output(OUTPUT_FILE_Q1B, v_out);
}


/** Main
 *
 * Set up CUDA device, read input file, and run Q1a and Q1b.
 *
 * \param	argc	Number of command-line arguments
 * \param	argv	Array of command-line arguments
 * \return	Program return code
 */
int main (int argc, char **argv) {
	#if DEBUG
	std::printf("Executing main...\n");
	#endif

	std::vector<int> v_in;
	int device_count;
	int dev = 0;
	hipDeviceProp_t dev_props;

	#if DEBUG
	printf("Detecting CUDA devices...\n");
	#endif

	// Check there are CUDA devices available
	hipGetDeviceCount(&device_count);
	if (device_count == 0) {
		fprintf(stderr, "ERROR:main: no CUDA devices found\n");
		exit(EXIT_FATAL);
	}

	// Use device 0
	hipSetDevice(dev);

	if (hipGetDeviceProperties(&dev_props, dev) == 0) {
		#if DEBUG
		printf("Using device:\n"
			   "\tID: %d\n"
			   "\tName: %s\n"
			   "\tGlobal mem: %d B\n"
			   "\tMax threads per block: %d\n"
			   "\tCompute: v%d.%d\n"
			   "\tClock: %d kHz\n",
			   dev,
			   dev_props.name,
			   (int)dev_props.totalGlobalMem,
			   (int)dev_props.maxThreadsPerBlock,
			   (int)dev_props.major,
			   (int)dev_props.minor,
			   (int)dev_props.clockRate);
		#endif
	} else {
		fprintf(stderr, "ERROR:main: could not find CUDA device information\n");
		exit(EXIT_FATAL);
	}

	#if DEBUG
	std::printf("Reading input array...\n");
	#endif

	// Read input array
	v_in = read_input(INPUT_FILE);

	#if DEBUG >= 2
	printf("\tInput array = [ ");
	for (int i=0; i<v_in.size(); ++i) {
		if (i == v_in.size()-1) {
			printf("%d ]\n", v_in[i]);
		} else {
			printf("%d, ", v_in[i]);
		}
	}
	#endif

	#if DEBUG
	std::printf("Running Q1 a...\n");
	#endif

	// Problem q1 a
	q1a(v_in, &dev_props);

	#if DEBUG
	std::printf("Running Q1 b...\n");
	#endif

	// Problem q1 b
	q1b(v_in, &dev_props);

	#if DEBUG
	std::printf("Done\n");
	#endif

	return 0;
}
